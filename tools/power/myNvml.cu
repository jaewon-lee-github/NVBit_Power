#include "hip/hip_runtime.h"
#include <cstdlib>
#include "cbt.h"
#include "myNvml.h"
#include <sys/time.h>

using namespace std;
template <typename T>
void check(T result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
                static_cast<unsigned int>(result), nvmlErrorString(result), func);
        nvmlShutdown();
        exit(EXIT_FAILURE);
    }
}
#define checkCudaNvmlErrors(val) check((val), #val, __FILE__, __LINE__)

myNvml::myNvml(int device, int sampling_interval, int reset_interval, int freq_mode, int bin_policy, int min_freq, int max_freq, int step_freq)
{
    const char *envVarValue = std::getenv("BENCH_NAME");
    debug_printf("BENCH_NAME= %s\n", envVarValue);
    if (envVarValue != NULL)
    {
        strncpy(bench_name, envVarValue, sizeof(bench_name));
    }
    else
    {
        strncpy(bench_name, "unknown", sizeof(bench_name));
    }
    _freq_mode = freq_mode;
    _bin_policy = bin_policy;
    target_device = device;
    start_flag = 0;
    CBT = new CallBackTimer();
    _min_freq = min_freq;
    _max_freq = max_freq;
    _step_freq = step_freq;
    _sampling_interval = sampling_interval;
    _reset_interval = reset_interval;
    bm = new BinManager(min_freq, max_freq, step_freq, sampling_interval, reset_interval);
    num_call = 0;
    prev_energy = 0;
    prev_power = 0;
    prev_avg_power = 0;
    total_power = 0;
    isFixed = false;
}
myNvml::~myNvml()
{
    // hipEventDestroy(start);
    // hipEventDestroy(stop);
    delete CBT;
    delete bm;
}

void myNvml::measure_init()
{
    // hipEventRecord(start);
}

void myNvml::measure_fin()
{
    debug_printf("Measure fin\n");
    fclose(ofile);
    checkCudaNvmlErrors(nvmlDeviceResetGpuLockedClocks(device));
    nvmlShutdown();
}

// We will measured power only once for kernel
//

void myNvml::reset()
{
    num_call = 0;
    prev_energy = 0;
    prev_power = 0;
    total_power = 0;
    prev_avg_power = 0;
}

void myNvml::measure_start(const char *k_name)
{
    if (start_flag == 0)
    {
        char temp[256];
        start_flag++;
        debug_printf("Measure start\n");
        debug_printf("freq_mode: %d\n", _freq_mode);
        checkCudaNvmlErrors(nvmlInit());
        checkCudaNvmlErrors(nvmlDeviceGetHandleByIndex(target_device, &device));
        sprintf(temp, "output_%d_%d_%d_%d_%d_%d_%s_%d_%d.csv",
                target_device, _freq_mode, _bin_policy, _min_freq, _max_freq, _step_freq,
                bench_name, _sampling_interval, _reset_interval);
        ofile = fopen(temp, "w");
        // otfile = fopen("time.csv", "w");
        reset();
        fprintf(ofile, "Benchmark,Kernel,Timestamp,Freq,FreqMode,BinPolicy,Power\n");
        // fseek(ofile, -1, SEEK_CUR);
    }
    if (_bin_policy == 10) // FIXME
    {
        strncpy(kernel_name, k_name, sizeof(kernel_name));
        debug_printf("Measure the power of kernel %s\n", kernel_name);
        CBT->start(_sampling_interval, [this](void)
                   { this->measure_energy_thread(); });
    }
    else
    {
        if (kernel_map.find(k_name) == kernel_map.end())
        {
            strncpy(kernel_name, k_name, sizeof(kernel_name));
            debug_printf("Newly executed kernel(%s) will be measured for power\n", kernel_name);
            reset();
            kernel_map[kernel_name] = 1;
            CBT->start(_sampling_interval, [this](void)
                       { this->measure_energy_thread(); });
        }
        else
        {
            debug_printf("Power of Kernel(%s) is already measured \n", k_name);
        }
    }
}

void myNvml::measure_stop()
{
    debug_printf("measure_stop(%s)\n", __func__);
    CBT->stop();
    // float milliseconds = 0;
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // fprintf(otfile, "%f\n", milliseconds);
    // measure_fin();
}

void myNvml::measure_energy_thread()
{
    // unsigned long long energy = 0;
    // double powerDiff = 0;
    unsigned int gpu_clock = 0;
    // Get GPU clock and change GPU clock
    checkCudaNvmlErrors(nvmlDeviceGetClockInfo(device, NVML_CLOCK_GRAPHICS, &gpu_clock));
    //  Get power Usage
    unsigned int powerUsage = 0;
    checkCudaNvmlErrors(nvmlDeviceGetPowerUsage(device, &powerUsage));
    total_power += powerUsage;

    // unsigned long long energy = 0;
    // checkCudaNvmlErrors(nvmlDeviceGetTotalEnergyConsumption(device, &energy));
    // powerDiff = ((double)energy - (double)prev_energy) / (double)sampling_interval;
    // if (powerDiff == 0)
    //     powerDiff = prev_power;
    // else
    // {
    //     prev_energy = energy;
    //     prev_power = powerDiff;
    // }
    // debug_printf("Total energy consumed: %llu\n", energy);
    // debug_printf("power from energy: %lf\n", powerDiff);

    // random bin frequency mode
    if (_freq_mode == FREQ_MODE::ORG)
    {
        // Nothing
    }
    // else if (_freq_mode == FREQ_MODE::FIXED && isFixed == false)
    else if (_freq_mode == FREQ_MODE::FIXED)
    {
        unsigned int target_freq = bm->getFreq();
        debug_printf("Freq will be fixed to %u\n", target_freq);
        checkCudaNvmlErrors(nvmlDeviceSetGpuLockedClocks(device, target_freq, target_freq));
        // isFixed = true;
    }
    else
    {
        debug_printf("[%d][%d] Current clock: %d\n", num_call, _freq_mode, gpu_clock);
        debug_printf("Power usage: %u\n", powerUsage);
        // num_call = 0 @init time
        if (num_call % bm->getResetPeriod() == 0)
        {
            debug_printf("Reset period\n");
            unsigned long long avg_power = total_power / bm->getResetPeriod();
            if (_freq_mode == FREQ_MODE::RANDOM)
                bm->setBinCounters(BIN_POLICY::FLAT);
            else if (_freq_mode == FREQ_MODE::ADAPTIVE)
            {
                if (avg_power < prev_avg_power) // prev_avg_power == 0 @init
                {
                    debug_printf("BIN_POLICY = INCLINED (%llu < %llu)\n", avg_power, prev_avg_power);
                    bm->setBinCounters(BIN_POLICY::INCLINED);
                }
                else
                {
                    debug_printf("BIN_POLICY = DECLINED (%llu >= %llu)\n", avg_power, prev_avg_power);
                    bm->setBinCounters(BIN_POLICY::DECLINED);
                }
            }
            prev_avg_power = avg_power;
            total_power = 0;
        }
        unsigned int target_freq = bm->getFreq();
        debug_printf("Freq will be changed to %u\n", target_freq);
        checkCudaNvmlErrors(nvmlDeviceSetGpuLockedClocks(device, target_freq, target_freq));
    }

    // Kernel,Timestamp,Freq,Power
    fprintf(ofile, "%s,%s,%u,%u,%u,%u,%u\n", bench_name, kernel_name, num_call, gpu_clock, _freq_mode, _bin_policy, powerUsage);
    // fprintf(ofile, "%u,%u,%f", gpu_clock, powerUsage, powerDiff);
    prev_power = powerUsage;
    num_call++;
}
